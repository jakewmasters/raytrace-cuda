#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <zlib.h>

#include "sphere.cpp"
#include "hitablelist.cpp"
#include "float.h"
#include "camera.cpp"

float hit_sphere(const vec3& center, float radius, const ray& r){
    vec3 oc = r.origin() - center;
    float a = dot(r.direction(), r.direction());
    float b = 2.0 * dot(oc, r.direction());
    float c = dot(oc, oc) - radius*radius;
    float discriminant = b*b - 4*a*c;
    if(discriminant < 0){
        return -1.0;
    } else {
        return (-b - sqrt(discriminant)) / (2.0*a);
    }
}

vec3 color(const ray& r, hitable *world/*, int depth*/){
    hit_record rec;
    if (world->hit(r,0.001,MAXFLOAT,rec)) {
        vec3 target = rec.p + rec.normal + random_in_unit_sphere();
        return 0.5*color(ray(rec.p, target-rec.p),world);
        // ray scattered;
        // vec3 attenuation;
        // if (depth < 50 && rec.mat_ptr->scatter(r, rec, attenuation, scattered)) {
        //     return attenuation*color(scattered, world, depth+1);
        // } else {
        //     return vec3(0,0,0);
        // }
    } else {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5*(unit_direction.y() + 1.0);
        return (1.0-t)*vec3(1.0,1.0,1.0) + t*vec3(0.5,0.7,1.0);    
    }
}

void usage(char *prog_name) {
    fprintf(stderr, "%s: [-h] -o <out_file>...\n", prog_name);
    fprintf(stderr, "  -o        exactly one output file must be specified\n");
    fprintf(stderr, "  -h        print this help and exit\n");
}

int main(int argc, char **argv) {

    char *prog_name = argv[0];
    char *out_file;

    int ch;
    while ((ch = getopt(argc, argv, "ho:")) != -1) {
        switch(ch) {
            case 'o':
                out_file = optarg;
                break;
            case '?':
            default:
                usage(prog_name);
        }
    }

    int nx = 512;
    int ny = 512;
    int ns = 100;

    FILE *out_file_ptr;
    if ((out_file_ptr = fopen(out_file, "w")) == NULL) {
        fprintf(stderr, "can't open %s for reading...\n", out_file);
        exit(1);
    }
    fprintf(out_file_ptr, "P3\n%d %d\n255\n", nx, ny);

    hitable *list[4];
    list[0] = new sphere(vec3(0,0,-1), 0.5, new lambertian(vec3(0.8,0.3,0.3)));
    list[1] = new sphere(vec3(0,-100.5,-1), 100, new lambertian(vec3(0.8,0.8,0.0)));
    list[2] = new sphere(vec3(1,0,-1),0.5, new metal(vec3(0.8,0.6,0.2)));
    list[3] = new sphere(vec3(-1,0,-1),0.5, new metal(vec3(0.8,0.8,0.8)));
    hitable *world = new hitable_list(list,2);
    camera cam;

    for (int j=ny-1; j >=0; --j){
        for (int i=0; i < nx; ++i){
            vec3 col(0,0,0);
            for (int s=0; s < ns; ++s){
                float u = float(i + drand48()) / float(nx);
                float v = float(j + drand48()) / float(ny);
                ray r = cam.get_ray(u,v);
                vec3 p = r.point_at_parameter(2.0);
                col += color(r,world/*, 0*/);    
            }
            col /= float(ns);
            col = vec3(sqrt(col[0]), sqrt(col[1]), sqrt(col[2]));
            int ir = int(255.99*col[0]);
            int ig = int(255.99*col[1]);
            int ib = int(255.99*col[2]);
            fprintf(out_file_ptr, "%d %d %d\n", ir, ig, ib);
        }
    }
}